/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLKSIZEX        16
#define BLKSIZEY        16

__global__
void GDFTgen_grid_kernel(double *pbecke, double *coords, double *atm_coords, double *a, int ngrids, int natm)
{
    int atom_i = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_id = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (grid_id >= ngrids || atom_i >= natm) {
        return;
    }

    double dx, dy, dz;
    double xi = atm_coords[3*atom_i+0];
    double yi = atm_coords[3*atom_i+1];
    double zi = atm_coords[3*atom_i+2];

    double xg = coords[3*grid_id+0];
    double yg = coords[3*grid_id+1];
    double zg = coords[3*grid_id+2];

    // distance between grids and atom i
    dx = xg - xi;
    dy = yg - yi;
    dz = zg - zi;
    double dig = sqrt(dx*dx + dy*dy + dz*dz);
    double becke = 2.0;
    for (int atom_j = 0; atom_j < natm; atom_j++){
        double xj = atm_coords[3*atom_j+0];
        double yj = atm_coords[3*atom_j+1];
        double zj = atm_coords[3*atom_j+2];

        // distance between grids and atom j
        dx = xg - xj;
        dy = yg - yj;
        dz = zg - zj;
        double djg = sqrt(dx*dx + dy*dy + dz*dz);
    
        // distance between atom i and atom j
        dx = xi - xj;
        dy = yi - yj;
        dz = zi - zj;
        double dij = sqrt(dx*dx + dy*dy + dz*dz);
        double g = (atom_i == atom_j) ? 0.0 : (dig - djg) / dij;
    
        // atomic radii adjust function
        double g1 = g*g;
        g1 -= 1.0;
        g1 *= a[atom_i * natm + atom_j];
        g += g1;

        // becke scheme
        g = (3.0 - g*g) * g * .5;
        g = (3.0 - g*g) * g * .5;
        g = (3.0 - g*g) * g * .5;

        g = 0.5 * (1.0 - g);
        becke *= g;
    }
    pbecke[atom_i*ngrids + grid_id] = becke;
}

extern "C"{
__host__
int GDFTgen_grid_partition(hipStream_t stream, double *pbecke, double *coords, double *atm_coords, double *a, int ngrids, int natm)
{
    dim3 threads(BLKSIZEX, BLKSIZEY);
    dim3 blocks((natm+BLKSIZEX-1)/BLKSIZEX, (ngrids+BLKSIZEY-1)/BLKSIZEY);
    GDFTgen_grid_kernel<<<blocks, threads, 0, stream>>>(pbecke, coords, atm_coords, a, ngrids, natm);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "CUDA Error of gen grids: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
    }
}