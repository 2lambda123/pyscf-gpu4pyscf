/* Copyright 2024 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 128

// (n,ncart,stride) -> (n,nsph,stride), count = n*stride
__global__
static void _cart2sph_ang2(double *cart, double *sph, int stride, int count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count){
        return;
    }
    int i = idx / stride;
    int j = idx % stride;
    int sph_offset = 5 * stride * i + j;
    int cart_offset = 6 * stride * i + j;
    double g0 = cart[cart_offset+0*stride];
    double g1 = cart[cart_offset+1*stride];
    double g2 = cart[cart_offset+2*stride];
    double g3 = cart[cart_offset+3*stride];
    double g4 = cart[cart_offset+4*stride];
    double g5 = cart[cart_offset+5*stride];

    sph[sph_offset+0*stride] = 1.092548430592079070 * g1;
    sph[sph_offset+1*stride] = 1.092548430592079070 * g4;
    sph[sph_offset+2*stride] = 0.630783130505040012 * g5 - 0.315391565252520002 * (g0 + g3);
    sph[sph_offset+3*stride] = 1.092548430592079070 * g2;
    sph[sph_offset+4*stride] = 0.546274215296039535 * (g0 - g3);
}

__global__
static void _cart2sph_ang3(double *cart, double *sph, int stride, int count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count){
        return;
    }
    int i = idx / stride;
    int j = idx % stride;
    int sph_offset = 7 * stride * i + j;
    int cart_offset = 10 * stride * i + j;
    double g0 = cart[cart_offset+0*stride];
    double g1 = cart[cart_offset+1*stride];
    double g2 = cart[cart_offset+2*stride];
    double g3 = cart[cart_offset+3*stride];
    double g4 = cart[cart_offset+4*stride];
    double g5 = cart[cart_offset+5*stride];
    double g6 = cart[cart_offset+6*stride];
    double g7 = cart[cart_offset+7*stride];
    double g8 = cart[cart_offset+8*stride];
    double g9 = cart[cart_offset+9*stride];

    sph[sph_offset+0*stride] = 1.770130769779930531 * g1 - 0.590043589926643510 * g6;
    sph[sph_offset+1*stride] = 2.890611442640554055 * g4;
    sph[sph_offset+2*stride] = 1.828183197857862944 * g8 - 0.457045799464465739 * (g1 + g6);
    sph[sph_offset+3*stride] = 0.746352665180230782 * g9 - 1.119528997770346170 * (g2 + g7);
    sph[sph_offset+4*stride] = 1.828183197857862944 * g5 - 0.457045799464465739 * (g0 + g3);
    sph[sph_offset+5*stride] = 1.445305721320277020 * (g2 - g7);
    sph[sph_offset+6*stride] = 0.590043589926643510 * g0 - 1.770130769779930530 * g3;
}

__global__
static void _cart2sph_ang4(double *cart, double *sph, int stride, int count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count){
        return;
    }
    int i = idx / stride;
    int j = idx % stride;
    int sph_offset = 9 * stride * i + j;
    int cart_offset = 15 * stride * i + j;
    double g0 = cart[cart_offset+0*stride];
    double g1 = cart[cart_offset+1*stride];
    double g2 = cart[cart_offset+2*stride];
    double g3 = cart[cart_offset+3*stride];
    double g4 = cart[cart_offset+4*stride];
    double g5 = cart[cart_offset+5*stride];
    double g6 = cart[cart_offset+6*stride];
    double g7 = cart[cart_offset+7*stride];
    double g8 = cart[cart_offset+8*stride];
    double g9 = cart[cart_offset+9*stride];
    double g10 = cart[cart_offset+10*stride];
    double g11 = cart[cart_offset+11*stride];
    double g12 = cart[cart_offset+12*stride];
    double g13 = cart[cart_offset+13*stride];
    double g14 = cart[cart_offset+14*stride];

    sph[sph_offset+0*stride] = 2.503342941796704538 * (g1 - g6);
    sph[sph_offset+1*stride] = 5.310392309339791593 * g4 - 1.770130769779930530 * g11;
    sph[sph_offset+2*stride] = 5.677048174545360108 * g8 - 0.946174695757560014 * (g1 + g6);
    sph[sph_offset+3*stride] = 2.676186174229156671 * g13- 2.007139630671867500 * (g4 + g11);
    sph[sph_offset+4*stride] = 0.317356640745612911 * (g0 + g10) + 0.634713281491225822 * g3 - 2.538853125964903290 * (g5 + g12) + 0.846284375321634430 * g14;
    sph[sph_offset+5*stride] = 2.676186174229156671 * g9 - 2.007139630671867500 * (g2 + g7);
    sph[sph_offset+6*stride] = 2.838524087272680054 * (g5 - g12) + 0.473087347878780009 * (g10 - g0);
    sph[sph_offset+7*stride] = 1.770130769779930531 * g2 - 5.310392309339791590 * g7 ;
    sph[sph_offset+8*stride] = 0.625835735449176134 * (g0 + g10) - 3.755014412695056800 * g3;
}

extern "C" {
__host__
int cart2sph(hipStream_t stream, double *cart_gto, double *sph_gto, int stride, int count, int ang)
{
    dim3 threads(THREADS);
    dim3 blocks((count + THREADS - 1)/THREADS);
    switch (ang) {
        case 0: break;
        case 1: break;
        case 2: _cart2sph_ang2 <<<blocks, threads, 0, stream>>> (cart_gto, sph_gto, stride, count); break;
        case 3: _cart2sph_ang3 <<<blocks, threads, 0, stream>>> (cart_gto, sph_gto, stride, count); break;
        case 4: _cart2sph_ang4 <<<blocks, threads, 0, stream>>> (cart_gto, sph_gto, stride, count); break;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
