/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <hipsolver.h>

extern "C" {
__host__
int eigh(hipsolverHandle_t cusolverH, double *a, double *b, double *w, int n)
{
    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1;     // A*x = (lambda)*B*x
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverStatus_t status;

    int lda = n;
    int lwork = 0;

    int *info;
    double *work = nullptr;
    
    // cache it?
    hipMalloc(reinterpret_cast<void **>(&info), sizeof(int));
    status = hipsolverDnDsygvd_bufferSize(cusolverH, itype, jobz, uplo, n, a, lda, b, lda, w, &lwork);
    if(status != HIPSOLVER_STATUS_SUCCESS){
        return 1;
    }
    hipMalloc(reinterpret_cast<void **>(&work), sizeof(double) * lwork);
    status = hipsolverDnDsygvd(cusolverH, itype, jobz, uplo, n, a, lda, b, lda, w, work, lwork, info);
    if(status != HIPSOLVER_STATUS_SUCCESS){
        return 1;
    }
     /* free resources */
    hipFree(info);
    hipFree(work);

    return 0;
}
}
