#include "hip/hip_runtime.h"

// general case, it is not supposed to be used in actual execution
template <int NROOTS, int GSIZE> __global__
static void GINTfill_int3c2e_ipvip1_kernel(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    fprintf(stderr, "general function is not implemented");
}

#if POLYFIT_ORDER_IP >= 2
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<2,GSIZE2_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[4];
    
    double g0[4*GSIZE2_INT3C];
    double *g1 = g0 + GSIZE2_INT3C;
    double *g2 = g1 + GSIZE2_INT3C;
    double *g3 = g2 + GSIZE2_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root2(x, uw);
            GINTscale_u<2>(uw, theta);
            GINTg0_2e_2d4d<2>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<2>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<2>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<2>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<2>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 3
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<3,GSIZE3_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[6];
    
    double g0[4*GSIZE3_INT3C];
    double *g1 = g0 + GSIZE3_INT3C;
    double *g2 = g1 + GSIZE3_INT3C;
    double *g3 = g2 + GSIZE3_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root3(x, uw);
            GINTscale_u<3>(uw, theta);
            GINTg0_2e_2d4d<3>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<3>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<3>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<3>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<3>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 4
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<4,GSIZE4_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[8];
    
    double g0[4*GSIZE4_INT3C];
    double *g1 = g0 + GSIZE4_INT3C;
    double *g2 = g1 + GSIZE4_INT3C;
    double *g3 = g2 + GSIZE4_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root4(x, uw);
            GINTscale_u<4>(uw, theta);
            GINTg0_2e_2d4d<4>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<4>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<4>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<4>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<4>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 5
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<5,GSIZE5_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[10];
    
    double g0[4*GSIZE5_INT3C];
    double *g1 = g0 + GSIZE5_INT3C;
    double *g2 = g1 + GSIZE5_INT3C;
    double *g3 = g2 + GSIZE5_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root5(x, uw);
            GINTscale_u<5>(uw, theta);
            GINTg0_2e_2d4d<5>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<5>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<5>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<5>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<5>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 6
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<6,GSIZE6_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[12];
    
    double g0[4*GSIZE6_INT3C];
    double *g1 = g0 + GSIZE6_INT3C;
    double *g2 = g1 + GSIZE6_INT3C;
    double *g3 = g2 + GSIZE6_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root6(x, uw);
            GINTscale_u<6>(uw, theta);
            GINTg0_2e_2d4d<6>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<6>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<6>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<6>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<6>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 7
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<7,GSIZE7_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[14];
    
    double g0[4*GSIZE7_INT3C];
    double *g1 = g0 + GSIZE7_INT3C;
    double *g2 = g1 + GSIZE7_INT3C;
    double *g3 = g2 + GSIZE7_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root7(x, uw);
            GINTscale_u<7>(uw, theta);
            GINTg0_2e_2d4d<7>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<7>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<7>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<7>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<7>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 8
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<8,GSIZE8_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[16];
    
    double g0[4*GSIZE8_INT3C];
    double *g1 = g0 + GSIZE8_INT3C;
    double *g2 = g1 + GSIZE8_INT3C;
    double *g3 = g2 + GSIZE8_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root8(x, uw);
            GINTscale_u<8>(uw, theta);
            GINTg0_2e_2d4d<8>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<8>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<8>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<8>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<8>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 

#if POLYFIT_ORDER_IP >= 9
template <> __global__
void GINTfill_int3c2e_ipvip1_kernel<9,GSIZE9_INT3C>(GINTEnvVars envs, ERITensor eri, BasisProdOffsets offsets)
{
    int ntasks_ij = offsets.ntasks_ij;
    int ntasks_kl = offsets.ntasks_kl;
    int task_ij = blockIdx.x * blockDim.x + threadIdx.x;
    int task_kl = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (task_ij >= ntasks_ij || task_kl >= ntasks_kl) {
        return;
    }
    int bas_ij = offsets.bas_ij + task_ij;
    int bas_kl = offsets.bas_kl + task_kl;
    double norm = envs.fac;
    double omega = envs.omega;
    int nprim_ij = envs.nprim_ij;
    int nprim_kl = envs.nprim_kl;
    int prim_ij = offsets.primitive_ij + task_ij * nprim_ij;
    int prim_kl = offsets.primitive_kl + task_kl * nprim_kl;
    int *bas_pair2bra = c_bpcache.bas_pair2bra;
    int *bas_pair2ket = c_bpcache.bas_pair2ket;
    int ish = bas_pair2bra[bas_ij];
    int jsh = bas_pair2ket[bas_ij];
    int ksh = bas_pair2bra[bas_kl];
    int lsh = bas_pair2ket[bas_kl];
    double uw[18];
    
    double g0[4*GSIZE9_INT3C];
    double *g1 = g0 + GSIZE9_INT3C;
    double *g2 = g1 + GSIZE9_INT3C;
    double *g3 = g2 + GSIZE9_INT3C;
    double* __restrict__ exp_bra = c_bpcache.a1;
    double* __restrict__ exp_ket = c_bpcache.a2;
    double* __restrict__ a12 = c_bpcache.a12;
    double* __restrict__ x12 = c_bpcache.x12;
    double* __restrict__ y12 = c_bpcache.y12;
    double* __restrict__ z12 = c_bpcache.z12;
    int ij, kl;
    int as_ish, as_jsh, as_ksh, as_lsh;
    if (envs.ibase) {
        as_ish = ish;
        as_jsh = jsh;
    } else {
        as_ish = jsh;
        as_jsh = ish;
    }
    if (envs.kbase) {
        as_ksh = ksh;
        as_lsh = lsh;
    } else {
        as_ksh = lsh;
        as_lsh = ksh;
    }
    for (ij = prim_ij; ij < prim_ij+nprim_ij; ++ij) {
        for (kl = prim_kl; kl < prim_kl+nprim_kl; ++kl) {
            double aij = a12[ij];
            double xij = x12[ij];
            double yij = y12[ij];
            double zij = z12[ij];
            double akl = a12[kl];
            double xkl = x12[kl];
            double ykl = y12[kl];
            double zkl = z12[kl];
            double xijxkl = xij - xkl;
            double yijykl = yij - ykl;
            double zijzkl = zij - zkl;
            double aijkl = aij + akl;
            double a1 = aij * akl;
            double a0 = a1 / aijkl;
            double theta = omega > 0.0 ? omega * omega / (omega * omega + a0) : 1.0; 
            a0 *= theta;
            double x = a0 * (xijxkl * xijxkl + yijykl * yijykl + zijzkl * zijzkl);
            GINTrys_root9(x, uw);
            GINTscale_u<9>(uw, theta);
            GINTg0_2e_2d4d<9>(envs, g0, uw, norm, as_ish, as_jsh, as_ksh, as_lsh, ij, kl);
            
            double ai2 = -2.0*exp_bra[ij];
            double aj2 = -2.0*exp_ket[ij];
            GINTnabla1j_2e<9>(envs, g1, g0, aj2, envs.i_l+1, envs.j_l, envs.k_l);
            GINTnabla1i_2e<9>(envs, g2, g0, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTnabla1i_2e<9>(envs, g3, g1, ai2, envs.i_l,   envs.j_l, envs.k_l);
            GINTwrite_int3c2e_ipip_direct<9>(envs, eri, g0, g1, g2, g3, ish, jsh, ksh);
    } }
}
#endif 
